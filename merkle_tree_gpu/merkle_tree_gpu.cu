#include "hip/hip_runtime.h"
#include "../merkle_tree.hpp"
#include "../cuda_hash_lib/md5.cuh"
#include "../cuda_hash_lib/sha256.cuh"
#include <cassert>
#include <cmath>
#include <openssl/sha.h>
#include <openssl/md5.h>
using namespace std;

int BLOCK_SIZE = 1024;

// hash algorithms - GPU versions
// SHA256
SHA_256_GPU::SHA_256_GPU() {
  digest_size = SHA256_DIGEST_LENGTH;
}

void SHA_256_GPU::get_hash(unsigned char* data,
                            int data_len,
                            unsigned char* hash) {
  SHA256(data, data_len, hash);
}

void SHA_256_GPU::get_hash(unsigned char* din,
                            int block_size,
                            unsigned char* dout,
                            int num_of_blocks) {
  int threadsPerBlock = 1024;
  int numOfBlocks = ceil(double(num_of_blocks)/ threadsPerBlock);
  dim3 dimGrid(numOfBlocks);
  dim3 dimBlock(threadsPerBlock);
  kernel_sha256_hash<<<dimGrid, dimBlock>>>(din, block_size,
                                            dout, num_of_blocks);
}

// MD5
MD_5_GPU::MD_5_GPU() {
  digest_size = MD5_DIGEST_LENGTH;
}

void MD_5_GPU::get_hash(unsigned char* data,
                        int data_len,
                        unsigned char* hash) {
  MD5(data, data_len, hash);
}

void MD_5_GPU::get_hash(unsigned char* din,
                        int block_size,
                        unsigned char* dout,
                        int num_of_blocks) {
  int threadsPerBlock = 1024;
  int numOfBlocks = ceil(double(num_of_blocks)/ threadsPerBlock);
  dim3 dimGrid(numOfBlocks);
  dim3 dimBlock(threadsPerBlock);
  kernel_md5_hash<<<dimGrid, dimBlock>>>(din, block_size,
                                         dout, num_of_blocks);
}


string hash_to_hex_string(unsigned char *hash, int size) {
  char temp[3];
  string result = "";
  for (int i = 0; i < size; i++) {
    snprintf(temp, 3, "%02x", hash[i]);
    result += temp;
  }
  return result;
}

void hex_string_to_hash(string hash_str, unsigned char* hash, int size) {
  int hash_str_size = hash_str.size();
  if (hash_str_size / 2 > size) {
    return;
  }
  unsigned char buf;
  for (int i = 0; i < hash_str_size; i += 2) {
    sscanf(hash_str.c_str() + i, "%02hhx", &buf);
    hash[i / 2] = buf;
  }
}

SHA_256::SHA_256() {
  digest_size = SHA256_DIGEST_LENGTH;
}

void SHA_256::get_hash(unsigned char* data,
                        int data_len,
                        unsigned char* hash) {
  SHA256(data, data_len, hash);
}


MD_5::MD_5() {
  digest_size = MD5_DIGEST_LENGTH;
}

void MD_5::get_hash(unsigned char* data,
                    int data_len,
                    unsigned char* hash) {
  MD5(data, data_len, hash);
}


//
// Class Block
//
Block::Block() {
  data = (unsigned char *)calloc(BLOCK_SIZE, sizeof(unsigned char));
}

//
// Class Blocks
//

// Blocks destructor
// TODO(allenpintsung): make this more smart with Rule of Five
Blocks::~Blocks() {
  for (auto &block : _blocks) {
    delete (block.data);
  }
}
vector<Block> const &Blocks::blocks() { return _blocks; }
Blocks::Blocks(unsigned char *data, int data_len) {
  int num_of_blocks = data_len / BLOCK_SIZE;
  int offset = 0;
  for (int i = 0; i < num_of_blocks; i++) {
    Block b;
    memcpy(b.data, data + offset, BLOCK_SIZE);
    offset += BLOCK_SIZE;
    _blocks.push_back(b);
  }
  if (offset < data_len) {
    Block b;
    memcpy(b.data, data + offset, data_len - offset);
    _blocks.push_back(b);
  }
}
void Blocks::add_blocks(Blocks &new_blocks) {
  _blocks.insert(_blocks.end(), new_blocks.blocks().begin(),
                 new_blocks.blocks().end());
}

//
// class MerkleNode
//
MerkleNode::MerkleNode()
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      hash(nullptr), digest_len(0) {}

MerkleNode::MerkleNode(unsigned char* hash_, int digest_len_)
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      hash(nullptr), digest_len(digest_len_) {
  hash = new unsigned char[digest_len];
  memset(hash, 0, digest_len);
  memcpy(hash, hash_, digest_len);
}

// make a MerkleNode with a specific hash_str
MerkleNode::MerkleNode(string hash_str, Hasher* hasher)
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      digest_len(hasher->hash_length()) {
  assert(hash_str.size() == digest_len * 2);
  hash = (unsigned char*)calloc(digest_len, sizeof(unsigned char));
  hex_string_to_hash(hash_str, hash, digest_len);
}

// make a MerkleNode from a block
MerkleNode::MerkleNode(const Block &block, Hasher* hasher)
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      digest_len(hasher->hash_length()) {
  hash = (unsigned char*)calloc(digest_len, sizeof(unsigned char));
  hasher->get_hash(block.data, BLOCK_SIZE, hash);
}

// make a parent MerkleNode from two child MerkleNodes (lhs, rhs)
MerkleNode::MerkleNode(MerkleNode *lhs, MerkleNode *rhs, Hasher* hasher)
    : parent(nullptr), lr(NA), digest_len(hasher->hash_length()) {
  hash = (unsigned char*)calloc(digest_len, sizeof(unsigned char));
  unsigned char* data =
      (unsigned char *)calloc(digest_len * 2, sizeof(unsigned char));
  memcpy(data, lhs->hash, digest_len);
  memcpy(data + digest_len, rhs->hash, digest_len);
  hasher->get_hash(data, digest_len * 2, hash);
  left = lhs;
  right = rhs;
  lhs->parent = this; // connect parent
  rhs->parent = this;
  lhs->lr = LEFT; // indicate left or right child
  rhs->lr = RIGHT;
}

// make a parent MerkleNode from an existing MerkleNode and its siblings,
// with info of left or right indicator.
MerkleNode::MerkleNode(MerkleNode cur_node, MerkleNode *sibling, Hasher* hasher)
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      digest_len(hasher->hash_length()) {
  hash = (unsigned char*)calloc(digest_len, sizeof(unsigned char));
  unsigned char* data =
      (unsigned char *)calloc(digest_len * 2, sizeof(unsigned char));
  if (sibling->lr == LEFT) {
    memcpy(data, sibling->hash, digest_len);
    memcpy(data + digest_len, cur_node.hash, digest_len);
  } else {
    memcpy(data, cur_node.hash, digest_len);
    memcpy(data + digest_len, sibling->hash, digest_len);
  }
  hasher->get_hash(data, digest_len * 2, hash);
}

// make a parent MerkleNode from an existing MerkleNode and its siblings,
// with info of left or right indicator.
MerkleNode::MerkleNode(MerkleNode cur_node, MerkleNode sibling, Hasher* hasher)
    : parent(nullptr), left(nullptr), right(nullptr), lr(NA),
      digest_len(hasher->hash_length()) {
  hash = (unsigned char*)calloc(digest_len, sizeof(unsigned char));
  unsigned char* data =
      (unsigned char *)calloc(digest_len * 2, sizeof(unsigned char));
  if (sibling.lr == LEFT) {
    memcpy(data, sibling.hash, digest_len);
    memcpy(data + digest_len, cur_node.hash, digest_len);
  } else {
    memcpy(data, cur_node.hash, digest_len);
    memcpy(data + digest_len, sibling.hash, digest_len);
  }
  hasher->get_hash(data, digest_len * 2, hash);
}

// print the hash of a MerkleNode in hex string format
void MerkleNode::print_hash() {
  for (int i = 0; i < digest_len; i++) {
    printf("%02x", hash[i]);
  }
  cout << endl;
}

// print the infomation of a MerkleNode
void MerkleNode::print_info() {
  string parent_hash;
  if (parent != nullptr){
    parent_hash = hash_to_hex_string(parent->hash, digest_len);
  } else {
    parent_hash = "";
  }
  cout << "parent hash: " << parent_hash << endl;
  cout << "l or r: " << lr << endl;
  cout <<   hash_to_hex_string(hash, digest_len) << endl;
}

//
// Class MerkleTree
//
void MerkleTree::delete_tree_walker(MerkleNode *cur_node) {
  if (cur_node == nullptr) {
    return;
  }
  delete_tree_walker(cur_node->left);
  delete_tree_walker(cur_node->right);
  delete (cur_node);
}

// produce a MerkleTree from hashes
MerkleNode *
MerkleTree::make_tree_from_hashes(vector<MerkleNode *>& cur_layer_nodes) {
  int cur_layer_nodes_size = cur_layer_nodes.size();
  while (cur_layer_nodes_size > 1) {
    cur_layer_nodes_size = cur_layer_nodes.size();
    int count = 0;
    for (int i = 0; i < cur_layer_nodes_size - 1; i = i + 2) {
      cur_layer_nodes[count] =
          new MerkleNode(cur_layer_nodes[i], cur_layer_nodes[i + 1], hasher);
      count++;
    }
    if (count > 0 && cur_layer_nodes_size % 2 != 0) {
      cur_layer_nodes[count] = cur_layer_nodes[cur_layer_nodes_size - 1];
      cur_layer_nodes.resize(count + 1);
    } else {
      cur_layer_nodes.resize(count);
    }
  }
  assert(cur_layer_nodes[0]->parent == nullptr);
  return cur_layer_nodes[0];
}

// produce a MerkleTree from Blocks and assign the head to root
MerkleNode *MerkleTree::make_tree_from_blocks(Blocks &blocks) {
  if (blocks.blocks().empty()) {
    return nullptr;
  }
  vector<MerkleNode *> cur_layer_nodes;
  for (const auto &block : blocks.blocks()) {
    MerkleNode *to_add = new MerkleNode(block, hasher);
    cur_layer_nodes.push_back(to_add);
    string hash_str = hash_to_hex_string(to_add->hash, hasher->hash_length());
    hashes.push_back(to_add);
    hash_leaf_map[hash_str] = to_add;
  }
  return make_tree_from_hashes(cur_layer_nodes);
}

// helper functions in verification process
bool MerkleTree::verify(MerkleNode cur_node, vector<MerkleNode *> &siblings) {
  for (const auto &sibling : siblings) {
    cur_node = MerkleNode(cur_node, sibling, hasher);
  }
  if (memcmp(cur_node.hash, root->hash, hasher->hash_length()) == 0) {
    return true;
  } else {
    return false;
  }
}

// print a MerkleTree, layer by layer, left to right
void MerkleTree::print() {
  queue<MerkleNode *> q;
  q.push(root);
  int layer = 0;
  while (!q.empty() && q.front() != nullptr) {
    cout << "Layer " << layer << ":" << endl;
    int size = q.size();
    while (size > 0) {
      auto node = q.front();
      q.pop();
      node->print_hash();
      if (node->left != nullptr) {
        q.push(node->left);
      }
      if (node->right != nullptr) {
        q.push(node->right);
      }
      size--;
    }
    layer++;
  }
}

// return a string contains the root hash of the MerkleTree in hex string format
string MerkleTree::root_hash() {
  return hash_to_hex_string(root->hash, hasher->hash_length());
}

// print the root hash in hex string format
void MerkleTree::print_root_hash() { cout << root_hash() << endl; }

// constructor with only Hasher
MerkleTree::MerkleTree(Hasher* hasher_) : hasher(hasher_) {}

// constructor using Blocks
MerkleTree::MerkleTree(Blocks& blocks_, Hasher* hasher_) : hasher(hasher_) {
  root = make_tree_from_blocks(blocks_);
}

// constructor using data in unsigned char and data_len
MerkleTree::MerkleTree(unsigned char* data, int data_len, Hasher* hasher_)
    : hasher(hasher_) {
  // Blocks blocks(data, data_len);
  // root = make_tree_from_blocks(blocks);

  // Note(allenpthuang): for temporary testing
  // SHA_256_GPU gpu_hasher;
  int num_of_blocks = (data_len % BLOCK_SIZE) ? data_len / BLOCK_SIZE + 1 : data_len / BLOCK_SIZE;
  int in_bytes = num_of_blocks * BLOCK_SIZE;
  int out_bytes = num_of_blocks * hasher->hash_length();

  unsigned char *out = (unsigned char *)calloc(out_bytes, sizeof(unsigned char));
  unsigned char *dout, *din;
  hipMalloc((void**) &dout, out_bytes);
  hipMalloc((void**) &din, in_bytes);
  hipMemset(din, 0, in_bytes);
  hipMemcpy(din, data, data_len, hipMemcpyHostToDevice);
  hasher->get_hash(din, BLOCK_SIZE, dout, num_of_blocks);
  // kernel_sha256_hash<<<1, num_of_blocks>>>(din, BLOCK_SIZE, dout, num_of_blocks);
  hipMemcpy(out, dout, out_bytes, hipMemcpyDeviceToHost);
  hipFree(dout); hipFree(din);

  // out has all hashes
  vector<MerkleNode *> cur_layer_nodes;
  for (int i = 0; i < num_of_blocks; ++i) {
    string hash_str = hash_to_hex_string(out + i * hasher->hash_length(),
                                         hasher->hash_length());
    MerkleNode *to_add = new MerkleNode(hash_str, hasher);
    cur_layer_nodes.push_back(to_add);
    hashes.push_back(to_add);
    hash_leaf_map[hash_str] = to_add;
  }

  root = make_tree_from_hashes(hashes);
}

int next_pow_of_2(int input) {
  int r = 0;
  while (input >>= 1) {
    r++;
  }
  return r;
}

// Further acceleration
MerkleTree::MerkleTree(unsigned char* data, int data_len, Hasher* hasher_,
                       unsigned short accel_mask)
    : hasher(hasher_) {
  if ((accel_mask & NO_ACCEL) == NO_ACCEL) {
    Blocks blocks(data, data_len);
    root = make_tree_from_blocks(blocks);
    return;
  }
  int num_of_blocks = (data_len % BLOCK_SIZE) ? data_len / BLOCK_SIZE + 1 : data_len / BLOCK_SIZE;
  int in_bytes = num_of_blocks * BLOCK_SIZE;
  // TODO(allenpthuang): need to know how to calc out_bytes
  int out_bytes = num_of_blocks * hasher->hash_length() * 2 * 2;
  // int out_bytes = pow(2, next_pow_of_2(num_of_blocks)) * hasher->hash_length() * 2;

  unsigned char *out = (unsigned char *)calloc(out_bytes, sizeof(unsigned char));
  unsigned char *dout, *din;
  hipMalloc((void**) &dout, out_bytes);
  hipMalloc((void**) &din, in_bytes);

  if (! (din && dout)) {
    cerr << "Error allocating device memory for din and dout!" << endl;
    exit(1);
  }

  unsigned int *dparents, *dlefts, *drights;
  LeftOrRightSib *dlrs;
  if ((accel_mask & ACCEL_LINK) == ACCEL_LINK) {
    // TODO(allenpthuang): ditto, need to know how to calc.
    arr_size = num_of_blocks * 2 * 2;
    parents = (unsigned int *)calloc(arr_size, sizeof(unsigned int));
    lefts = (unsigned int *)calloc(arr_size, sizeof(unsigned int));
    rights = (unsigned int *)calloc(arr_size, sizeof(unsigned int));
    lrs = (LeftOrRightSib *)calloc(arr_size, sizeof(LeftOrRightSib));
    hipMalloc((void**) &dparents, arr_size * sizeof(unsigned int));
    hipMalloc((void**) &dlefts, arr_size * sizeof(unsigned int));
    hipMalloc((void**) &drights, arr_size * sizeof(unsigned int));
    hipMalloc((void**) &dlrs, arr_size * sizeof(LeftOrRightSib));
    if (! (dparents && dlefts && drights && dlrs)) {
      cerr << "Error allocating device memory for din and dout!" << endl;
      exit(1);
    }
  }

  hipMemset(din, 0, in_bytes);
  hipMemcpy(din, data, data_len, hipMemcpyHostToDevice);

  unsigned char *dout_left;
  unsigned char *dout_right = dout;

  hasher->get_hash(din, BLOCK_SIZE, dout, num_of_blocks);

  if ((accel_mask & (ACCEL_CREATION | ACCEL_REDUCTION))
        == (ACCEL_CREATION | ACCEL_REDUCTION)) {
    bool attached = false;
    for (auto n = num_of_blocks; n > 0; n /= 2) {
      dout_left = dout_right;
      if (attached) {
        n += 1;
        attached = false;
      }
      dout_right += n * hasher->hash_length();

      int threadsPerBlock = 1024;
      int numOfBlocks = ceil(double(n / 2) / threadsPerBlock);
      dim3 dimGrid(numOfBlocks);
      dim3 dimBlock(threadsPerBlock);
      if ((accel_mask & ACCEL_LINK) == ACCEL_LINK) {
        kernel_sha256_hash_link<<<dimGrid, dimBlock>>>(dout_left,
                                                       hasher->hash_length(),
                                                       dout_right,
                                                       n / 2,
                                                       dout,
                                                       dparents,
                                                       dlefts,
                                                       drights,
                                                       dlrs);
      } else {
        kernel_sha256_hash_cont<<<dimGrid, dimBlock>>>(dout_left,
                                                       hasher->hash_length(),
                                                       dout_right,
                                                       n / 2);
      }
      if (n / 2 > 0 && n % 2 != 0) {
        unsigned char *attach_pos = dout_right + (n / 2) * hasher->hash_length();
        unsigned char *copy_pos = dout_left + (n - 1) * hasher->hash_length();
        hipMemcpy(attach_pos, copy_pos, hasher->hash_length(),
                  hipMemcpyDeviceToDevice);
        attached = true;
      }
    }

    hipMemcpy(out, dout, out_bytes, hipMemcpyDeviceToHost);
    hipFree(dout); hipFree(din);

    if ((accel_mask & ACCEL_LINK) == ACCEL_LINK) {
      hipMemcpy(parents, dparents,
                 arr_size * sizeof(unsigned long), hipMemcpyDeviceToHost);
      hipMemcpy(lefts, dlefts,
                 arr_size * sizeof(unsigned long), hipMemcpyDeviceToHost);
      hipMemcpy(rights, drights,
                 arr_size * sizeof(unsigned long), hipMemcpyDeviceToHost);
      hipMemcpy(lrs, dlrs,
                 arr_size * sizeof(LeftOrRightSib), hipMemcpyDeviceToHost);
      hipFree(dparents);
      hipFree(dlefts);
      hipFree(drights);
      hipFree(dlrs);
    }

    unsigned char *result_out = out + (dout_right - dout) - hasher->hash_length();
    MerkleNode* root_node = new MerkleNode(result_out, hasher->hash_length());
    root = root_node;
    return;
  }

  if ((accel_mask & ACCEL_CREATION) == ACCEL_CREATION) {
    hipMemcpy(out, dout, out_bytes, hipMemcpyDeviceToHost);
    hipFree(dout);
    hipFree(din);

    // out has all hashes
    vector<MerkleNode *> cur_layer_nodes;
    for (int i = 0; i < num_of_blocks; ++i) {
      string hash_str = hash_to_hex_string(out + i * hasher->hash_length(),
                                           hasher->hash_length());
      MerkleNode *to_add = new MerkleNode(hash_str, hasher);
      cur_layer_nodes.push_back(to_add);
      hashes.push_back(to_add);
      hash_leaf_map[hash_str] = to_add;
    }

    root = make_tree_from_hashes(hashes);
    return;
  }
}

// delete the MerkleTree
void MerkleTree::delete_tree() {
  delete_tree_walker(root);
  root = nullptr;
}

// TODO(allenpthuang): Naive way to append blocks! Should be more efficient.
void MerkleTree::append(Blocks &new_blocks) {
  for (const auto& block : new_blocks.blocks()) {
    MerkleNode* to_add = new MerkleNode(block, hasher);
    hashes.push_back(to_add);
  }
  delete_tree();
  root = make_tree_from_hashes(hashes);
}

void MerkleTree::append(unsigned char* data, int data_len) {
  Blocks blocks_to_append(data, data_len);
  append(blocks_to_append);
}

// return a vector of the pointer to the sibling MerkleNodes along
// the path to the root.
vector<MerkleNode *> MerkleTree::find_siblings(MerkleNode *leaf) {
  vector<MerkleNode *> result;
  MerkleNode *cur_node = leaf;
  while (cur_node->parent != nullptr) {
    if (cur_node->lr == LEFT) {
      result.push_back(cur_node->parent->right);
    } else {
      result.push_back(cur_node->parent->left);
    }
    cur_node = cur_node->parent;
  }
  return result;
}

// return a vector of the sibling MerkleNodes along the path to the root.
vector<MerkleNode> MerkleTree::find_siblings(string hash_str) {
  MerkleNode *cur_node;
  auto it = hash_leaf_map.find(hash_str);
  if (it != hash_leaf_map.end()) {
    cur_node = it->second;
  } else {
    return {};
  }

  vector<MerkleNode> result;
  while (cur_node != nullptr && cur_node->parent != nullptr) {
    MerkleNode tmp;
    if (cur_node->lr == LEFT) {
      tmp = (*cur_node->parent->right);
    } else {
      tmp = (*cur_node->parent->left);
    }
    result.push_back(tmp);
    cur_node = cur_node->parent;
  }
  return result;
}

// verify whether a piece of data exists in the MerkleTree
bool MerkleTree::verify(unsigned char *data, int data_len) {
  Blocks blocks_to_verify(data, data_len);
  for (auto block : blocks_to_verify.blocks()) {
    if (!verify(block)) {
      return false;
    }
  }
  return true;
}

// verify whether a block of data exists in the MerkleTree
bool MerkleTree::verify(Block &block) {
  unsigned char* hash =
      (unsigned char*)calloc(hasher->hash_length(), sizeof(unsigned char));
  hasher->get_hash(block.data, BLOCK_SIZE, hash);
  return verify(hash_to_hex_string(hash, hasher->hash_length()));
}

// verify whether a hash_str of some data exists in the MerkleTree
bool MerkleTree::verify(string hash_str) {
  if (hash_str.size() != hasher->hash_length() * 2) {
    return false;
  }
  if (hash_leaf_map.find(hash_str) == hash_leaf_map.end()) {
    return false;
  }
  MerkleNode *node = hash_leaf_map[hash_str];
  MerkleNode input = (*node);
  auto siblings = find_siblings(node);
  return verify(input, siblings);
}


// verify whether a hash_str of some data exists in the MerkleTree,
// using only sibling MerkleNodes and the root hash.
bool MerkleTree::verify(string hash_str, vector<MerkleNode> &siblings,
                        string root_hash) {
  MerkleNode cur_node(hash_str, hasher);
  for (auto &sibling : siblings) {
    sibling.print_hash();
    cur_node = MerkleNode(cur_node, sibling, hasher);
  }
  string calculated = hash_to_hex_string(cur_node.hash, hasher->hash_length());
  cout << "check root hash" << endl;
  if (calculated == root_hash) {
    return true;
  } else {
    return false;
  }
}
