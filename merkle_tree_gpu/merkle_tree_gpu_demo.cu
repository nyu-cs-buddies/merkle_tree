#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <unistd.h>
#include "../merkle_tree.hpp"

using namespace std;

void print_node(MerkleNode* node) {
  cout << hash_to_hex_string(node->hash, node->digest_len) << endl;
}

int main(int argc, char *argv[]) {
  // Hasher can be SHA_256 or MD_5 at the moment.
  Hasher* hasher = new SHA_256_GPU();

  BLOCK_SIZE = 1024;
  unsigned char* data;
  int data_len = 0;
  if (argc == 1) {
    // no input file; use dummy data for demo.
    cerr << "Usage: ./merkle_tree_gpu_demo <BLOCK_SIZE> <filename>" << endl;
    cerr << "For demo, create data filled with '9527' with " << BLOCK_SIZE * 4
         << " bytes." << endl;
    data = (unsigned char *)malloc(BLOCK_SIZE * 4 * sizeof(unsigned char));
    data_len = BLOCK_SIZE * 4;
    memset(data + BLOCK_SIZE * 0, 9, BLOCK_SIZE);
    memset(data + BLOCK_SIZE * 1, 5, BLOCK_SIZE);
    memset(data + BLOCK_SIZE * 2, 2, BLOCK_SIZE);
    memset(data + BLOCK_SIZE * 3, 7, BLOCK_SIZE);
  } else if (argc == 3) {
    BLOCK_SIZE = atoi(argv[1]);
    // input filepath provided
    ifstream is;
    is.open(argv[2], ios::binary | ios::ate);
    if (! is.good()) {
      cerr << "File not found at: " << argv[2] << endl;
      exit(2);
    }
    // show file info and read into a buffer
    data_len = is.tellg();
    cout << "path = " << argv[2] << endl;
    cout << "filesize = " << data_len << endl;

    data = (unsigned char *)malloc(data_len * sizeof(unsigned char));
    is.clear();
    is.seekg(0, std::ios::beg);
    is.read((char*)data, data_len);
  } else {
    cerr << "Usage: ./merkle_tree_gpu_demo <BLOCK_SIZE> <filename>" << endl;
    cerr << "Or ./merkle_tree_gpu_demo to demo with dummy data." << endl;
    exit(1);
  }

  // make Blocks from data for further demo
  Blocks blocks(data, data_len);

  // make a MerkleTree from data
  // MerkleTree merkle_tree(data, data_len, hasher);
  unsigned short ACCEL_MASK = ACCEL_CREATION | ACCEL_REDUCTION | ACCEL_LINK | ACCEL_HASHMAP;
  MerkleTree merkle_tree(data, data_len, hasher, ACCEL_MASK);
  cout << "===== Read all at once. =====" << endl;
  cout << "BLOCK_SIZE = " << BLOCK_SIZE << endl;
  merkle_tree.print();
  cout << "Root hash: ";
  merkle_tree.print_root_hash();

  // cout << "=========== P/L/R pointers testing zone =============" << endl;

  // MerkleNode* root = merkle_tree.root;
  // cout << "Root" << endl;
  // print_node(root);
  // cout << "Left child" << endl;
  // print_node(root->left);
  // cout << "Right child" << endl;
  // print_node(root->right);
  // cout << "Right child's parent!" << endl;
  // print_node(root->right->parent);
  // cout << "Right child's right child!" << endl;
  // print_node(root->right->right);
  // cout << "Right child's right child's parent!" << endl;
  // print_node(root->right->right->parent);

  // cout << "=========== P/L/R ends ==============================" << endl;

  int block_idx = 0;
  auto block_to_verify = blocks.blocks()[0];
  printf("===== Test Block #%d out of %lu blocks =====\n", block_idx + 1,
         blocks.blocks().size());

  // if (merkle_tree.verify(block_to_verify.data, BLOCK_SIZE)) {
  //   cout << "Yeah! Verified!" << endl;
  // }
  if (merkle_tree.verify(block_to_verify)) {
    cout << "Yeah! Verified!" << endl;
  }

  cout << "==== Verify as a client ====" << endl;
  unsigned char* client_hash =
      (unsigned char*)calloc(hasher->hash_length(), sizeof(unsigned char));
  hasher->get_hash(block_to_verify.data, BLOCK_SIZE, client_hash);
  string hash_str = hash_to_hex_string(client_hash, hasher->hash_length());
  string root_hash = merkle_tree.root_hash();
  cout << "hash_str of the block: " << hash_str << endl;
  cout << "root_hash: " << root_hash << endl;
  auto siblings = merkle_tree.find_siblings(hash_str);
  MerkleTree local_tree(hasher);
  if (local_tree.verify(hash_str, siblings, root_hash)) {
    cout << "Yeah! Verified!" << endl;
  }

  /*

  // split input data into two halves; the second half is appended later.
  int num_of_blocks = ceil((double)data_len / BLOCK_SIZE);
  if (num_of_blocks > 1) {
    int first_size = BLOCK_SIZE * (num_of_blocks / 2);
    Blocks old_blocks(data, first_size);
    MerkleTree merkle_tree_to_append(old_blocks);
    cout << "===== Read half first, and append the other half. =====" << endl;
    cout << "=== Merkle Tree of the first half ===" << endl;
    merkle_tree_to_append.print();

    Blocks new_blocks(data + first_size, data_len - first_size);
    merkle_tree_to_append.append(new_blocks);
    cout << "=== Merkle Tree of the first half + the second half ===" << endl;
    merkle_tree_to_append.print();
  }

  */
  
  delete hasher;

  return 0;
}
